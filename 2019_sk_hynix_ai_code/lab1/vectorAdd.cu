#include<stdio.h>
#include<hip/hip_runtime.h>

/**
 * CUDA kernel code
 */
__global__
void vectorAdd(float *A,  float *B, float *C, int numElemnets)
{
	int i = threadIdx.x+blockDim.x*blockIdx.x;
	//vector addition
	if(i<numElemnets){
		C[i] = A[i] + B[i];
	}
}
/**
 * Host main routine
 */
int main(void)
{
	hipError_t err = hipSuccess;

	int n = 50000;
	size_t size = n * sizeof(float);
	// alloc host side memory
	float *h_A = (float*)malloc(size);
	float *h_B = (float*)malloc(size);
	float *h_C = (float*)malloc(size);

	//alloc device vetors
	float *d_A = NULL;
	float *d_B = NULL;
	float *d_C = NULL;
  //TODO: cudaMalloc for d_A, d_B, d_C
	hipMalloc(&d_A,size);
	hipMalloc(&d_B,size);
	hipMalloc(&d_C,size);

	//init vector A and vector B
	for(int j=0;j < n; j++){
	 h_A[j] = rand()%2;
	 h_B[j] = rand()%2;
	}

	// copy host data to device
	printf("Copy input vectors to device\n");
  //TODO: cudaMemcpy h_A -> d_A, h_B -> d_B
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);


	//Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads \n", blocksPerGrid, threadsPerBlock);
	vectorAdd<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,d_C,n);
	err = hipGetLastError();
	//error check
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//Copy device output data to host
	printf("Copy output data to host\n");
  //TODO: cudaMemcpy d_C -> h_C
	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

	//Verifiy output
	int pass = 0;

	pass = 1;
	for (int i=0;i<n;i++)
	{
		if(fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
		{
			pass = 0;
			fprintf(stderr, "Result is invalid at element %d!\n",i);
			exit(EXIT_FAILURE);
		}
	}

	if (pass)
		printf("Test PASSED\n");
	else
		printf("Test FAILED\n");

	//free device memory
  //TODO: cudaFree for d_A, d_B, d_C
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	//free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	printf("Done\n");
	return 0;
}
